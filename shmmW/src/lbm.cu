#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "definitions.h"
#include "geometry.h"

#define GPU_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{

    if (code != hipSuccess)
    {
        fprintf(stderr,"GPU ASSERT: %s %d %s %d\n", hipGetErrorString(code), code, file, line);
        if (abort) exit(code);
    }
}

__constant__ __device__
int offsets_device[2][6][2] =          /* Even/odd neighbor coordinate shifts */
    {
        { {0,1}, {1,0}, {1,-1}, {0,-1}, {-1,-1}, {-1,0} },
        { {0,1}, {1,1}, { 1,0}, {0,-1}, {-1, 0}, {-1,1} }
    };

__constant__ __device__
float force_device[2] = {0.0, FORCE};

__device__
int local_neighbor_x (int y, int x, int i, int* coords_device, int* dims_device) {
    if( GY_device(y-1)%2 ) return ( (x + offsets_device[1][i][1]));
    else      return ( (x + offsets_device[0][i][1]));
}

__device__
int local_neighbor_y (int y, int i, int* coords_device, int* dims_device) {
    if( GY_device(y-1)%2 ) return ( (y + offsets_device[1][i][0]));
    else      return ( (y + offsets_device[0][i][0]));
}

__global__ void __collide ( point_t* lattice_device_p,
                            bool* ghost_device_p,
                            float* c_device_p,
                            int* coords_device,
                            int* dims_device,
                            int local_grid_width_device,
                            int local_grid_height_device,
                            int local_grid_width_halo_device,
                            int local_grid_height_halo_device )
{

    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int x = blockDim.x * blockIdx.x + threadIdx.x;

    if (x >= local_grid_width_device || y >= local_grid_height_device) {
        return;
    }


    float rho = 0.0;
    float uc = 0.0;

#ifdef DEBUG
    if(LIB_device(x,y) >= local_grid_width_halo_device * local_grid_height_halo_device) {
        printf("Illegal x,y = %d,%d\n",x,y);
        printf("LIB(x,y) = %d\n", LIB_device(x,y));
        /* exit(1); */
    }
#endif // DEBUG

    /* Zero the velocity before computing it */
    lattice_device_p[LIB_device(x, y)].velocity[0] = lattice_device_p[LIB_device(x, y)].velocity[1] = 0.0;

    /* Compute velocity unless lattice site is a ghost */
    if ( ! ghost_device_p[GY_device(y) * L_WIDTH + GX_device(x)] )
    {
        for ( int i=0; i<6; i++ )
        {
            rho += lattice_device_p[LIB_device(x, y)].density[i][NOW];
            lattice_device_p[LIB_device(x, y)].velocity[0] += c_device_p[2*i] * lattice_device_p[LIB_device(x, y)].density[i][NOW];
            lattice_device_p[LIB_device(x, y)].velocity[1] += c_device_p[1 + 2*i] * lattice_device_p[LIB_device(x, y)].density[i][NOW];
        }
        /* rho*u = sum_i( Ni*ci ), so divide by rho to find u: */
        lattice_device_p[LIB_device(x, y)].velocity[0] /= rho;
        lattice_device_p[LIB_device(x, y)].velocity[1] /= rho;
    }

    for ( int i=0; i<6; i++ )
    {
        float
            qi_uaub,
            N_eq,
            delta_N;

        float c_device_p_0 = c_device_p[2*i];
        float c_device_p_1 = c_device_p[1 + 2*i];

        qi_uaub =
            ( c_device_p_1 * c_device_p_1 - 0.5 ) * lattice_device_p[LIB_device(x, y)].velocity[1] * lattice_device_p[LIB_device(x, y)].velocity[1] +
            ( c_device_p_1 * c_device_p_0       ) * lattice_device_p[LIB_device(x, y)].velocity[1] * lattice_device_p[LIB_device(x, y)].velocity[0] +
            ( c_device_p_0 * c_device_p_1       ) * lattice_device_p[LIB_device(x, y)].velocity[0] * lattice_device_p[LIB_device(x, y)].velocity[1] +
            ( c_device_p_0 * c_device_p_0 - 0.5 ) * lattice_device_p[LIB_device(x, y)].velocity[0] * lattice_device_p[LIB_device(x, y)].velocity[0];
        uc = lattice_device_p[LIB_device(x, y)].velocity[0] * c_device_p_0 + lattice_device_p[LIB_device(x, y)].velocity[1] * c_device_p_1;

        // Equilibrium, difference
        N_eq = ( rho / 6.0 ) * ( 1.0 + 2.0 * uc + 4.0 * qi_uaub );
        delta_N = LAMBDA * ( lattice_device_p[LIB_device(x, y)].density[i][NOW] - N_eq );

        // Apply external force at boundary
//        if ( GX_device(x)==1 )
        if ( FORCE_COND_CUDA )
            delta_N += (1.0/3.0) * (c_device_p_0*force_device[0] + c_device_p_1*force_device[1]);

        // Reflections at ghosts
        if( ! ghost_device_p[GY_device(y) * L_WIDTH + GX_device(x)] )
            lattice_device_p[LIB_device(x, y)].density[i][NEXT] = lattice_device_p[LIB_device(x, y)].density[i][NOW] + delta_N;
        else
            lattice_device_p[LIB_device(x, y)].density[(i+3)%6][NEXT] = lattice_device_p[LIB_device(x, y)].density[i][NOW];
    }



}

extern "C" void collide( point_t* lattice_device_p,
                         bool* ghost_device_p,
                         float* c_device_p,
                         int* coords_device,
                         int* dims_device,
                         int local_grid_width_device,
                         int local_grid_height_device,
                         int local_grid_width_halo_device,
                         int local_grid_height_halo_device )
{
    dim3 block(BLOCKSIZE, BLOCKSIZE);
    dim3 grid((local_grid_width_halo_device / BLOCKSIZE) + 1, (local_grid_height_halo_device / BLOCKSIZE) + 1);

    __collide<<<grid, block>>>(lattice_device_p,
                               ghost_device_p,
                               c_device_p,
                               coords_device,
                               dims_device,
                               local_grid_width_device,
                               local_grid_height_device,
                               local_grid_width_halo_device,
                               local_grid_height_halo_device );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Collide Error: %s\n", hipGetErrorString(err));
    hipDeviceSynchronize();

}

__global__
void
__propagate ( point_t* lattice_device_p, int* coords_device, int* dims_device, int local_grid_width_halo_device, int local_grid_height_halo_device, int iter, int rank)
{

    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int x = blockDim.x * blockIdx.x + threadIdx.x;

    if (x >= local_grid_width_halo_device || y >= local_grid_height_halo_device) {
        return;
    }

    for( int i=0; i<6; i++ )
    {
#ifdef DEBUG
        if(GX_device(x) == 0 && GY_device(y) == 0 && iter==0) {
            printf("RANK: %d -- ", rank);
            printf("local neighbor (i=%d), x=%d,y=%d\n",
                   i,
                   local_neighbor_x(y, x, i, coords_device, dims_device),
                   local_neighbor_y(y, i, coords_device, dims_device));
        }
#endif // DEBUG
        int n_x = local_neighbor_x(y, x, i, coords_device, dims_device);
        int n_y = local_neighbor_y(y, i, coords_device, dims_device);

#ifdef DEBUG
        if(n_x < -1 || n_y < -1) {
            printf("Illegal value n_x=%d,n_y=%d\n", n_x, n_y);
            /* exit(1); */
        }
#endif // DEBUG
        if (n_x <= -1 || n_y <= -1 || n_x >= local_grid_width_halo_device || n_y >= local_grid_height_halo_device) {
            continue;
        }

        /* if (ghost[GY(y)][GX(x)]) { */
        /*     continue; */
        /* } */

        /* if (ghost[GY(n_y)][GX(n_x)]) { */
        /*     continue; */
        /* } */

        lattice_device_p[LI_device(n_x, n_y)].density[i][NOW] = lattice_device_p[LI_device(x, y)].density[i][NEXT];



        // Used during debugging: no density should vanish from the system
        //    check_mass();

    }
}

extern "C" void propagate(point_t* lattice_device_p,
                          int* coords_device,
                          int* dims_device,
                          int local_grid_width_halo_device,
                          int local_grid_height_halo_device,
                          int iter,
                          int rank) {
    dim3 block(BLOCKSIZE, BLOCKSIZE);
    dim3 grid((local_grid_width_halo_device / BLOCKSIZE) + 1, (local_grid_height_halo_device / BLOCKSIZE) + 1);
    __propagate<<<grid, block>>>(lattice_device_p,
                                 coords_device,
                                 dims_device,
                                 local_grid_width_halo_device,
                                 local_grid_height_halo_device,
                                 iter,
                                 rank);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        printf("Propagate Error: %s\n", hipGetErrorString(err));
    hipDeviceSynchronize();
}

extern "C" void init_device(point_t** lattice_device,
                            point_t* lattice,
                            int local_grid_width_halo,
                            int local_grid_height_halo,
                            size_t sizeof_point_t,
                            bool** ghost_device,
                            bool* ghost,
                            float* c,
                            float** c_device,
                            int* coords,
                            int** coords_device,
                            int* dims,
                            int** dims_device) {
    GPU_CHECK(hipMalloc(lattice_device, local_grid_width_halo * local_grid_height_halo * sizeof_point_t));
    GPU_CHECK(hipMemcpy(*lattice_device, lattice, local_grid_width_halo * local_grid_height_halo * sizeof_point_t, hipMemcpyHostToDevice));

    GPU_CHECK(hipMalloc(ghost_device, L_HEIGHT * L_WIDTH * sizeof(bool)));
    GPU_CHECK(hipMemcpy(*ghost_device, ghost, L_HEIGHT*L_WIDTH*sizeof(bool), hipMemcpyHostToDevice));

    GPU_CHECK(hipMalloc(c_device, 2 * 6 * sizeof(float)));
    GPU_CHECK(hipMemcpy(*c_device, c, 2*6*sizeof(float), hipMemcpyHostToDevice));

    GPU_CHECK(hipMalloc(coords_device, NDIMS * sizeof(int)));
    GPU_CHECK(hipMemcpy(*coords_device, coords, NDIMS * sizeof(int), hipMemcpyHostToDevice));

    GPU_CHECK(hipMalloc(dims_device, NDIMS * sizeof(int)));
    GPU_CHECK(hipMemcpy(*dims_device, dims, NDIMS * sizeof(int), hipMemcpyHostToDevice));

}

extern "C" void copyToHost(point_t* lattice, point_t* lattice_device, int local_grid_width_halo, int local_grid_height_halo, int sizeof_point_t) {
    hipDeviceSynchronize();
    GPU_CHECK(hipMemcpy(lattice, lattice_device, local_grid_width_halo * local_grid_height_halo * sizeof_point_t, hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
}

extern "C" void copyToDevice(point_t* lattice, point_t* lattice_device, int local_grid_width_halo, int local_grid_height_halo, int sizeof_point_t) {
    hipDeviceSynchronize();
    GPU_CHECK(hipMemcpy(lattice_device, lattice, local_grid_width_halo * local_grid_height_halo * sizeof_point_t, hipMemcpyHostToDevice));
    hipDeviceSynchronize();
}

extern "C" void free_device(point_t* lattice_device,
                            float* c_device,
                            bool* ghost_device,
                            int* coords_device,
                            int* dims_device) {
    hipFree(lattice_device);
    hipFree(c_device);
    hipFree(ghost_device);
    hipFree(coords_device);
    hipFree(dims_device);
}
